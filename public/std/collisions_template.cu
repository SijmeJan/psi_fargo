#include "hip/hip_runtime.h"
#define __GPU
#define __NOPROTO

#include "fargo3d.h"

#define ymin(i) ymin_s[(i)]
CONSTANT(real, ymin_s, 3846);

__global__ void _collisions_kernel(real dt, 
				   int id1, 
				   int id2, 
				   int id3, 
				   int option,
				   int pitch,
				   int stride,
				   int size_x,
				   int size_y,
				   int size_z,
				   real *alpha,
				   %FLUIDS0) {

  int i;
  int j;
  int k;
  int ll;
  int o;
  int p;
  int q;
  int ir;
  int ir2;
  int ir_max;
  int ic;
  real max_value;
  real factor;
  real big;
  real temp;
  real sum;
  int idm;
  real b[NFLUIDS];
  real m[NFLUIDS*NFLUIDS];  
  real omega;
  real rho_p;
  real rho_o;
  real rho_q;

  %FLUIDS1;

#ifdef X 
  i = threadIdx.x + blockIdx.x * blockDim.x;
#else 
  i = 0;
#endif 
#ifdef Y 
  j = threadIdx.y + blockIdx.y * blockDim.y;
#else 
  j = 0;
#endif 
#ifdef Z 
  k = threadIdx.z + blockIdx.z * blockDim.z;
#else 
  k = 0;
#endif
  
#ifdef Z
  if(k>=1 && k<size_z) {
#endif
#ifdef Y
    if(j>=1 && j<size_y) {
#endif
#ifdef X
      if(i<size_x) {
#endif
	
#include  "collision_kernel.h"
#include  "gauss.h"
	
	for (o=0; o<NFLUIDS; o++) {
	  velocities_output[o][l] = b[o];
	}
	
#ifdef X 
      } 
#endif
#ifdef Y 
    } 
#endif
#ifdef Z 
  } 
#endif
}

extern "C" void _collisions_gpu(real dt, int id1, int id2, int id3, int option) {

  real *rho[NFLUIDS];
  real *velocities_input[NFLUIDS];
  real *velocities_output[NFLUIDS];

  int ii;

  for (ii=0; ii<NFLUIDS; ii++) {

    INPUT(Fluids[ii]->Density);
    rho[ii]  = Fluids[ii]->Density->field_gpu;
    
    //Collisions along X
#ifdef X
    if (id1 == 1) {
      if (option == 1) {
	INPUT(Fluids[ii]->Vx_temp);
	OUTPUT(Fluids[ii]->Vx_temp);
	velocities_input[ii] = Fluids[ii]->Vx_temp->field_gpu;
	velocities_output[ii] = Fluids[ii]->Vx_temp->field_gpu;
      }
      if (option == 0) {
	INPUT(Fluids[ii]->Vx);
	OUTPUT(Fluids[ii]->Vx_half);
	velocities_input[ii] = Fluids[ii]->Vx->field_gpu;
	velocities_output[ii] = Fluids[ii]->Vx_half->field_gpu;
      }
    }
#endif
    
    //Collisions along Y
#ifdef Y
    if (id2 == 1) {
      if (option == 1) {
	INPUT(Fluids[ii]->Vy_temp);
	OUTPUT(Fluids[ii]->Vy_temp);
	velocities_input[ii] = Fluids[ii]->Vy_temp->field_gpu;
	velocities_output[ii] = Fluids[ii]->Vy_temp->field_gpu;
      }
      if (option == 0) {
	INPUT(Fluids[ii]->Vy);
	OUTPUT(Fluids[ii]->Vy_half);
	velocities_input[ii] = Fluids[ii]->Vy->field_gpu;
	velocities_output[ii] = Fluids[ii]->Vy_half->field_gpu;
      }
    }
#endif
    
    //Collisions along Z
#ifdef Z
    if (id3 == 1) {
      if (option == 1) {
	INPUT(Fluids[ii]->Vz_temp);
	OUTPUT(Fluids[ii]->Vz_temp);
	velocities_input[ii] = Fluids[ii]->Vz_temp->field_gpu;
	velocities_output[ii] = Fluids[ii]->Vz_temp->field_gpu;
      }
      if (option == 0) {
	INPUT(Fluids[ii]->Vz);
	OUTPUT(Fluids[ii]->Vz_half);
	velocities_input[ii] = Fluids[ii]->Vz->field_gpu;
	velocities_output[ii] = Fluids[ii]->Vz_half->field_gpu;
      }
    }
#endif
  }
  
  dim3 block (BLOCK_X, BLOCK_Y, BLOCK_Z);
  dim3 grid ((Nx+2*NGHX+block.x-1)/block.x,
	     ((Ny+2*NGHY)+block.y-1)/block.y,
	     ((Nz+2*NGHZ)+block.z-1)/block.z);
  
#ifdef BIGMEM
#define ymin_d &Ymin_d
#endif

  CUDAMEMCPY(ymin_s, ymin_d, sizeof(real)*(Ny+2*NGHY+1), 0, hipMemcpyDeviceToDevice);

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(_collisions_kernel), hipFuncCachePreferL1 );

  _collisions_kernel<<<grid,block>>>(dt,
				     id1,
				     id2,
				     id3,
				     option,
				     Pitch_gpu,
				     Stride_gpu,
				     XIP,
				     Ny+2*NGHY,
				     Nz+2*NGHZ,
				     Alpha_d,
				     %FLUIDS2);

  check_errors("_collisions_kernel");

}
